#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include <iostream>
#include <assert.h>

#include <stdlib.h>

#include "option.h"
#include "nbody.h"

using namespace std;

#define THREAD_NUM 1024

class HIPContext
{
public:
    HIPContext();
    ~HIPContext();

    static void createContext();

    static int inited;
    static point *CudaCurrpoints;
    static point *CudaResults;
};

HIPContext::HIPContext()
{
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        cerr << "hipSetDevice failed! Do you have a CUDA-capable GPU installed?" << endl;
        exit(EXIT_FAILURE);
    }

    cudaStatus = hipMalloc((void **)&CudaCurrpoints, sizeof(point) * POINT_CNT);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        exit(EXIT_FAILURE);
    }

    cudaStatus = hipMalloc((void **)&CudaResults, sizeof(point) * POINT_CNT);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        exit(EXIT_FAILURE);
    }

    cout << "CUDA construct done" << endl;
}

HIPContext::~HIPContext()
{
    hipFree(CudaResults);
    hipFree(CudaCurrpoints);

    cout << "CUDA destruct done" << endl;
}

void HIPContext::createContext()
{
    assert(POINT_CNT < BLOCK_NUM *THREAD_NUM && "POINT_CNT is too large");
    static HIPContext context;
}

int HIPContext::inited = 0;
point *HIPContext::CudaCurrpoints = NULL;
point *HIPContext::CudaResults = NULL;

__global__ static void _nBodyCalculateCUDA(const point *currpoints, point *newpoints, double dt)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i >= POINT_CNT) {
        return;
    }

    float G = GRAVITATIONAL_G;
    float epi = 0.0000000000001;
    // Acceleration
    float ax = 0;
    float ay = 0;
    float az = 0;
    float sx = 0;
    float sy = 0;
    float sz = 0;
    float x = 0;
    float y = 0;
    float z = 0;

    #pragma unroll POINT_CNT
    for (int j = 0; j < POINT_CNT; ++j) {
        float rx;
        float ry;
        float rz;
        float lenpow2;
        float len;

        rx = currpoints[j]._x - currpoints[i]._x;
        ry = currpoints[j]._y - currpoints[i]._y;
        rz = currpoints[j]._z - currpoints[i]._z;

        lenpow2 = rx * rx + ry * ry + rz * rz + epi;
        len = sqrt(lenpow2);

        ax += currpoints[j]._mass * rx / len / lenpow2;
        ay += currpoints[j]._mass * ry / len / lenpow2;
        az += currpoints[j]._mass * rz / len / lenpow2;
    }

    ax *= G;
    ay *= G;
    az *= G;

    // Update speed
    sx = currpoints[i]._sx + ax * dt;
    sy = currpoints[i]._sy + ay * dt;
    sz = currpoints[i]._sz + az * dt;

    newpoints[i]._sx = sx;
    newpoints[i]._sy = sy;
    newpoints[i]._sz = sz;

    // Update position
    x = currpoints[i]._x + sx * dt;
    y = currpoints[i]._y + sy * dt;
    z = currpoints[i]._z + sz * dt;

    newpoints[i]._x = x;
    newpoints[i]._y = y;
    newpoints[i]._z = z;

    // Copy others property
    newpoints[i]._r = currpoints[i]._r;
    newpoints[i]._g = currpoints[i]._g;
    newpoints[i]._b = currpoints[i]._b;
    newpoints[i]._mass = currpoints[i]._mass;
    newpoints[i]._size = currpoints[i]._size;
}

void nBodyCalculateCUDA(const point *currpoints, point *newpoints, double dt)
{
    hipError_t cudaStatus;
    int block_num = (POINT_CNT + THREAD_NUM - 1) / THREAD_NUM;

    if (!HIPContext::inited) {
        HIPContext::createContext();
        HIPContext::inited = 1;
    }

    cudaStatus = hipMemcpy(HIPContext::CudaCurrpoints, currpoints, sizeof(point) * POINT_CNT, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        exit(EXIT_FAILURE);
    }

    _nBodyCalculateCUDA<<<block_num, THREAD_NUM>>>(HIPContext::CudaCurrpoints, HIPContext::CudaResults, dt);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        cerr << "addKernel launch failed: " << hipGetErrorString(cudaStatus) << endl;
        exit(EXIT_FAILURE);
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching addKernel!" << endl;
        exit(EXIT_FAILURE);
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(newpoints, HIPContext::CudaResults, sizeof(point) * POINT_CNT, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        exit(EXIT_FAILURE);
    }
}
#include "hip/hip_runtime.h"
#include <math.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#include "./header/nbody_cuda.cuh"

// #define BLOCK_SIZE 256

__global__ void _nBodyCalculateKernel(const point* currpoints, point* newpoints, double dt) {
    int thisI = threadIdx.x + blockIdx.x * blockDim.x;
	int thisJ = threadIdx.y + blockIdx.y * blockDim.y;
    if (thisI >= POINT_CNT || thisJ >= POINT_CNT) {
		// std::cout << "thisI, thisJ out of range : " << thisI << ' ' << thisJ << '\n';
		return;
	}
	else if(thisI == thisJ) return;
    point currentPoint = currpoints[thisI];
    point targetPoint = currpoints[thisJ];
    if (currentPoint._mass == 0.0f || targetPoint._mass == 0.0f) {
        newpoints[thisI] = currentPoint;
        return;
    }

    float G = GRAVITATIONAL_G;
    float epi = 0.0000000000001;
    float ax = 0, ay = 0, az = 0;
    float r_c = currentPoint._size / 2;
    float r_t = targetPoint._size / 2;

	bool check_x = false;
	bool check_y = false;
	bool check_z = false;
	float rfx = 1.0; // border reflect check
	float rfy = 1.0;
	float rfz = 1.0;

    // Copy current point to avoid race conditions

	float rx;
	float ry;
	float rz;
	float lenpow2;
	float len;

	float dis = r_c + r_t;

	rx = targetPoint._x - currentPoint._x;
	ry = targetPoint._y - currentPoint._y;
	rz = targetPoint._z - currentPoint._z;

	lenpow2 = rx * rx + ry * ry + rz * rz + epi;
	len = sqrt(lenpow2);

	// gravity
	ax += targetPoint._mass * rx / len / lenpow2;
	ay += targetPoint._mass * ry / len / lenpow2;
	az += targetPoint._mass * rz / len / lenpow2;	
    
	// check collapse
	if (len <= dis) {
		// collapse: 0, be eaten: 1, eat: 2
		int diff_char = currentPoint._character - targetPoint._character;
		if (diff_char < 0) diff_char += 3;

		// collapse
		if (diff_char == 0) {
			float sum_m = currentPoint._mass + targetPoint._mass;
			float diff_m = currentPoint._mass - targetPoint._mass;
			// Update speed
			newpoints[thisI]._sx = (diff_m * currentPoint._sx + 2 * targetPoint._mass * targetPoint._sx) / sum_m;
			newpoints[thisI]._sy = (diff_m * currentPoint._sy + 2 * targetPoint._mass * targetPoint._sy) / sum_m;
			newpoints[thisI]._sz = (diff_m * currentPoint._sz + 2 * targetPoint._mass * targetPoint._sz) / sum_m;
		}
		// be eaten
		else if (diff_char == 1) {
			newpoints[thisI]._mass = 0;
			newpoints[thisI]._size = 0;
		}
		// eat
		else if (diff_char == 2) {
			newpoints[thisI]._mass += targetPoint._mass;
			// M1/s1 = M2/s2, s2 = M2s1/M1
			newpoints[thisI]._size = newpoints[thisI]._mass * currentPoint._size / currentPoint._mass;

			newpoints[thisI]._sx = (currentPoint._sx * currentPoint._mass + targetPoint._sx * targetPoint._mass) / newpoints[thisI]._mass;
			newpoints[thisI]._sy = (currentPoint._sy * currentPoint._mass + targetPoint._sy * targetPoint._mass) / newpoints[thisI]._mass;
			newpoints[thisI]._sz = (currentPoint._sz * currentPoint._mass + targetPoint._sz * targetPoint._mass) / newpoints[thisI]._mass;
		}
	}

	ax *= G;
	ay *= G;
	az *= G;
	
	// check border
	if(currentPoint._x >= POINT_XYZ_MAX) {
		rfx = -1.0;
		check_x = true;
	}
	else if (currentPoint._x <= -1 * POINT_XYZ_MAX){
		rfx = 1.0;
		check_x = true;
	}

	if(currentPoint._y >= POINT_XYZ_MAX) {
		rfy = -1.0;
		check_y = true;
	}
	else if (currentPoint._y <= -1 * POINT_XYZ_MAX){
		rfy = 1.0;
		check_y = true;
	}

	if(currentPoint._z >= POINT_XYZ_MAX) {
		rfz = -1.0;
		check_z = true;
	}
	else if (currentPoint._z <= -1 * POINT_XYZ_MAX){
		rfz = 1.0;
		check_z = true;
	}

	// Update speed
	newpoints[thisI]._sx += ax * dt;
	newpoints[thisI]._sy += ay * dt;
	newpoints[thisI]._sz += az * dt;

	if (check_x) newpoints[thisI]._sx = abs(newpoints[thisI]._sx) * rfx; 
	if (check_y) newpoints[thisI]._sy = abs(newpoints[thisI]._sy) * rfy; 
	if (check_z) newpoints[thisI]._sz = abs(newpoints[thisI]._sz) * rfz; 

	// Update position
	newpoints[thisI]._x = currentPoint._x + newpoints[thisI]._sx * dt;
	newpoints[thisI]._y = currentPoint._y + newpoints[thisI]._sy * dt;
	newpoints[thisI]._z = currentPoint._z + newpoints[thisI]._sz * dt;	
}

void nBodyCalculateCUDA(const point* h_currpoints, point* h_newpoints, double dt) {
    point *d_currpoints, *d_newpoints;
    size_t size = POINT_CNT * sizeof(point);

    // Allocate device memory
    hipMalloc(&d_currpoints, size);
    hipMalloc(&d_newpoints, size);

    // Copy input arrays to device
    hipMemcpy(d_currpoints, h_currpoints, size, hipMemcpyHostToDevice);

    // Configure grid and block dimensions
    // int gridSize = (POINT_CNT + BLOCK_SIZE - 1) / BLOCK_SIZE;
	dim3 threadsPerBlock(32, 32);
    dim3 numBlocks((POINT_CNT + threadsPerBlock.x - 1) / (threadsPerBlock.x), // ceiling()
                (POINT_CNT + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Launch kernel
    _nBodyCalculateKernel<<<numBlocks, threadsPerBlock>>>(d_currpoints, d_newpoints, dt);

    // Copy result back to host
    hipMemcpy(h_newpoints, d_newpoints, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_currpoints);
    hipFree(d_newpoints);

    // Check for any CUDA errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        // std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
    }
}